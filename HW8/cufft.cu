#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <cmath>
#include <iostream>
#include <vector>
#include <chrono>

// 线性索引（stride=2）
__host__ __device__ inline size_t id3(int i, int j, int k, int n, int s)
{
    return ((size_t)i * n + j) * n + k;
}

// 频率索引
__host__ __device__ inline int fk(int idx, int n) { return (idx < n / 2 ? idx : idx - n); }

// CPU 端生成平面波
inline void makePlaneWave(std::vector<hipfftDoubleComplex> &A, int n, int stride)
{
    size_t M = (size_t)n * n * n * stride;
    for (size_t i = 0; i < M; i++)
    {
        A[i].x = 0;
        A[i].y = 0;
    }
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            for (int k = 0; k < n; k++)
            {
                int p = id3(i, j, k, n, stride);
                double phase = (i + j + k) * 1.0;
                A[p].x = cos(phase);
                A[p].y = sin(phase);
            }
        }
    }
}

// GPU 核：频域乘 i·k
__global__ void mulIk(const hipfftDoubleComplex *F, hipfftDoubleComplex *Gx,
                      hipfftDoubleComplex *Gy, hipfftDoubleComplex *Gz,
                      int n, int stride)
{
    int i = blockIdx.x;
    int j = blockIdx.y;
    int k = threadIdx.x;
    if (i < n && j < n && k < n)
    {
        int p = id3(i, j, k, n, stride);
        int kx = fk(i, n), ky = fk(j, n), kz = fk(k, n);
        double a = F[p].x, b = F[p].y;
        Gx[p].x = -kx * b;
        Gx[p].y = kx * a;
        Gy[p].x = -ky * b;
        Gy[p].y = ky * a;
        Gz[p].x = -kz * b;
        Gz[p].y = kz * a;
    }
}

int main(int argc, char *argv[])
{
    if (argc < 2)
        return 1;
    int n_trial = std::max(3, std::atoi(argv[1]));
    std::vector<int> n_list = {16, 32, 64, 128, 256};
    int stride = 2;
    std::cout << "n,avg_time_sec,flops\n";

    for (int n : n_list)
    {
        size_t N = (size_t)n * n * n;
        size_t M = N * stride;

        // 1) 在 CPU 上生成平面波并分配到 GPU
        std::vector<hipfftDoubleComplex> hA(M);
        makePlaneWave(hA, n, stride);

        hipfftDoubleComplex *dA, *dF, *dGx, *dGy, *dGz, *dgx, *dgy, *dgz;
        hipMalloc(&dA, M * sizeof(hipfftDoubleComplex));
        hipMalloc(&dF, M * sizeof(hipfftDoubleComplex));
        hipMalloc(&dGx, M * sizeof(hipfftDoubleComplex));
        hipMalloc(&dGy, M * sizeof(hipfftDoubleComplex));
        hipMalloc(&dGz, M * sizeof(hipfftDoubleComplex));
        hipMalloc(&dgx, M * sizeof(hipfftDoubleComplex));
        hipMalloc(&dgy, M * sizeof(hipfftDoubleComplex));
        hipMalloc(&dgz, M * sizeof(hipfftDoubleComplex));
        hipMemcpy(dA, hA.data(), M * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);

        // 2) 创建 cuFFT plan
        hipfftHandle p_fwd, p_inv;
        int dims[3] = {n, n, n};
        hipfftPlanMany(&p_fwd, 3, dims,
                      dims, stride, N,
                      dims, stride, N,
                      HIPFFT_Z2Z, 1);
        hipfftPlanMany(&p_inv, 3, dims,
                      dims, stride, N,
                      dims, stride, N,
                      HIPFFT_Z2Z, 1);

        float tot = 0;
        for (int t = 0; t < n_trial; t++)
        {
            // 复制初始数据
            hipMemcpy(dA, hA.data(), M * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);

            hipEvent_t st, ed;
            hipEventCreate(&st);
            hipEventCreate(&ed);
            hipEventRecord(st, 0);

            // 前向 FFT
            hipfftExecZ2Z(p_fwd, dA, dF, HIPFFT_FORWARD);

            // 频域上乘 i·k
            dim3 grid(n, n);
            dim3 block(n);
            mulIk<<<grid, block>>>(dF, dGx, dGy, dGz, n, stride);

            // 三次逆向 FFT
            hipfftExecZ2Z(p_inv, dGx, dgx, HIPFFT_BACKWARD);
            hipfftExecZ2Z(p_inv, dGy, dgy, HIPFFT_BACKWARD);
            hipfftExecZ2Z(p_inv, dGz, dgz, HIPFFT_BACKWARD);

            hipEventRecord(ed, 0);
            hipEventSynchronize(ed);
            float ms;
            hipEventElapsedTime(&ms, st, ed);
            tot += ms;

            hipEventDestroy(st);
            hipEventDestroy(ed);
        }
        double avg = tot / 1000.0 / n_trial;
        double flp = 20.0 * N * log2(N) + 24.0 * N;
        std::cout << n << "," << avg << "," << flp << "\n";

        hipfftDestroy(p_fwd);
        hipfftDestroy(p_inv);
        hipFree(dA);
        hipFree(dF);
        hipFree(dGx);
        hipFree(dGy);
        hipFree(dGz);
        hipFree(dgx);
        hipFree(dgy);
        hipFree(dgz);
    }
    return 0;
}